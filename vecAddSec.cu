#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "timers.h"

long int MAX;
__global__ void VecAdd(int* A, int* B, long int MAX)
{
  long int i = 0;
  while(i++ < MAX)
    A[i] += B[i];
}

// Host code
int main(int argc, char** argv)
{
  MAX = (long int)atoi(argv[1]);
  size_t size = MAX * sizeof(float);
  // Allocate input vectors h_A and h_B in host memory
  int* h_A = (int*)malloc(size);
  int* h_B = (int*)malloc(size);
  int i;
  pTimer zegar = newTimer();
  // Initialize input vectors
  srand(time(NULL));
  for(i=0;i<MAX;i++) {
    h_A[i] = rand();
    h_B[i] = rand();
  }
  
  // Allocate vectors in device memory
  int* d_A;
  hipMalloc(&d_A, size);
  int* d_B;
  hipMalloc(&d_B, size);
  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (MAX + threadsPerBlock -1) / threadsPerBlock;
  startTimer(zegar);
  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, MAX);
  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  stopTimer(zegar);
  printf("calkowity czas gpu sekwencyjnie: ");
  printTimer(zegar);
  printf("\n");
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  // Free host memory
  free(h_A);
  free(h_B);
  freeTimer(zegar);
}
