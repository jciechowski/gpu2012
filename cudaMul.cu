/*
Mnozenie macierzy CUDA,
Jakub Ciechowski GPU 2012

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



__global__ void naiveMul(int* M, int* N, int* P, int width) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int sum = 0;
    for(int k=0; k < width; k++) 
        sum += M[ty*width+k] * N[k*width+tx];

    P[ty*width+tx] = sum;
}
void printMat(int *a, int width) {
    int i,j;
    for(i=0;i<width;i++) 
        for(j=0;j<width;j++)
            printf("%d%c",a[i*width+j],(j == (width-1))?'\n':'\t');

    printf("\n");
}


int* genMatrix(int width) {
    int *a = (int*)calloc(width*width, sizeof(int));
    int i,j;
    for(i=0;i<width;i++)
        for(j=0;j<width;j++)
            a[i*width+j] = rand()%10;
    return a;
}


int *matMul(int *hostA, int *hostB, int width) {
    int SIZE = width*width;
    int *hostC = (int*)calloc(SIZE,sizeof(int));

    int *devA, *devB, *devC;
    hipMalloc((void**) &devA, SIZE*sizeof(int));
    hipMalloc((void**) &devB, SIZE*sizeof(int));
    hipMalloc((void**) &devC, SIZE*sizeof(int));

    dim3 blockDim(width, width);
    dim3 gridDim(1,1);

    hipMemcpy(devA, hostA, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devC, hostC, SIZE*sizeof(int), hipMemcpyHostToDevice);
    naiveMul<<<gridDim,blockDim>>>(devA, devB, devC, width);
    hipMemcpy(hostC, devC, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    return hostC;
}

int main(int argc, char** argv) {
    int m = 4;
    int *A;
    int *B;
    int *C;
    if(argc > 2) {
        m = atoi(argv[1]);
    }

    A = genMatrix(m);
    printMat(A,m);
    B = genMatrix(m);
    printMat(B,m);
    
    C = matMul(A,B,m);
    printMat(C,m);

    return 0;
}
