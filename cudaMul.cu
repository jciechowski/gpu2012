/*
Mnozenie macierzy CUDA,
Jakub Ciechowski GPU 2012

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_WIDTH 2

__global__ void sharedMul(int *M, int *N, int *P, int width) {

  __shared__ int Ms[TILE_WIDTH][TILE_WIDTH];
  __shared__ int Ns[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = by * TILE_WIDTH + ty;
  int col = bx * TILE_WIDTH + tx;

  int sum = 0;

  for(int m = 0; m < width/TILE_WIDTH; m++) {
    Ms[ty][tx] = M[row*width + (m*TILE_WIDTH + tx)];
    Ns[ty][tx] = N[(m*TILE_WIDTH + ty)*width + col];

    __syncthreads();

    for(int k = 0; k < TILE_WIDTH; k++) {
      sum += Ms[m+ty][k] * Ns[k][m+tx]; 
    }
    __syncthreads();
  }
  P[row*width+col] = sum; 
}


// __global__ void naiveMul(int* M, int* N, int* P, int width) {

//   // TODO 
//   // mnozenie macierzy powyzej 512 elementow
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;

//     int sum = 0;
//         sum += M[ty*width+k] * N[k*width+tx];

//     P[ty*width+tx] = sum;
// }

void printMat(int *a, int width) {
    int i,j;
    for(i=0;i<width;i++) 
        for(j=0;j<width;j++)
            printf("%d%c",a[i*width+j],(j == (width-1))?'\n':'\t');

    printf("\n");
}


int* genMatrix(int width) {
    int *a = (int*)calloc(width*width, sizeof(int));
    int i,j;
    for(i=0;i<width;i++)
        for(j=0;j<width;j++)
            a[i*width+j] = rand()%10;
    return a;
}


int *matMul(int *hostA, int *hostB, int width) {
    int SIZE = width*width;
    int *hostC = (int*)calloc(SIZE,sizeof(int));

    int *devA, *devB, *devC;
    hipMalloc((void**) &devA, SIZE*sizeof(int));
    hipMalloc((void**) &devB, SIZE*sizeof(int));
    hipMalloc((void**) &devC, SIZE*sizeof(int));

       dim3 gridDim(width/TILE_WIDTH, width/TILE_WIDTH);
       dim3 blockDim(TILE_WIDTH,TILE_WIDTH);
    // dim3 blockDim(width,width);
    // dim3 gridDim(1,1);

    hipMemcpy(devA, hostA, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devC, hostC, SIZE*sizeof(int), hipMemcpyHostToDevice);
       sharedMul<<<gridDim,blockDim>>>(devA, devB, devC, width);
    //    naiveMul<<<gridDim,blockDim>>>(devA, devB, devC, width);
    hipMemcpy(hostC, devC, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    return hostC;
}

int main(int argc, char** argv) {
    int m = 4;
    int *A;
    int *B;
    int *C;
    if(argc > 1) {
        m = atoi(argv[1]);
    }

    A = genMatrix(m);
    printMat(A,m);
    B = genMatrix(m);
    printMat(B,m);
    
    C = matMul(A,B,m);
    printf("TILE = %d\n",TILE_WIDTH);
    printMat(C,m);

    return 0;
}
