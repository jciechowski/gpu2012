/*
Mnozenie macierzy CUDA,
Jakub Ciechowski GPU 2012

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


int TILE_WIDTH = 5;

__global__ void tilingMul(int *M, int *N, int *P, int width, int TILE_WIDTH) {
  
  int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
  
  int sum = 0;
  
  for(int k = 0; k < width; k++)
    sum += M[row*width+k] * N[k*width+col];
  
  P[row*width+col] = sum;
  
}

__global__ void naiveMul(int* M, int* N, int* P, int width) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int sum = 0;
    for(int k=0; k < width; k++) 
        sum += M[ty*width+k] * N[k*width+tx];

    P[ty*width+tx] = sum;
}
void printMat(int *a, int width) {
    int i,j;
    for(i=0;i<width;i++) 
        for(j=0;j<width;j++)
            printf("%d%c",a[i*width+j],(j == (width-1))?'\n':'\t');

    printf("\n");
}

int* genMatrix(int width) {
    int *a = (int*)calloc(width*width, sizeof(int));
    int i,j;
    for(i=0;i<width;i++)
        for(j=0;j<width;j++)
            a[i*width+j] = rand()%10;
    return a;
}

int *matMul(int *hostA, int *hostB, int width) {
    int SIZE = width*width;
    int *hostC = (int*)calloc(SIZE,sizeof(int));

    int *devA, *devB, *devC;
    hipMalloc((void**) &devA, SIZE*sizeof(int));
    hipMalloc((void**) &devB, SIZE*sizeof(int));
    hipMalloc((void**) &devC, SIZE*sizeof(int));
    
    // do 256 elementow, naiveMul
//    dim3 blockDim(width, width);
//    dim3 gridDim(1,1);

    // powyzej 256
    TILE_WIDTH = sqrt(width);
    dim3 blockDim(width/TILE_WIDTH, width/TILE_WIDTH);
    dim3 gridDim(TILE_WIDTH, TILE_WIDTH);
    hipMemcpy(devA, hostA, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devB, hostB, SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devC, hostC, SIZE*sizeof(int), hipMemcpyHostToDevice);
    // do 256 elementow
//    naiveMul<<<gridDim,blockDim>>>(devA, devB, devC, width);
    // powyzej 256
    tilingMul<<<gridDim, blockDim>>> (devA, devB, devC, width, TILE_WIDTH);
    hipMemcpy(hostC, devC, SIZE*sizeof(int), hipMemcpyDeviceToHost);

    return hostC;
}

int main(int argc, char** argv) {
    int m = 4;
    int *A;
    int *B;
    int *C;
    if(argc > 2) {
        m = atoi(argv[1]);
    }

    A = genMatrix(m);
    printMat(A,m);
    B = genMatrix(m);
    printMat(B,m);
    
    C = matMul(A,B,m);
    printMat(C,m);

    return 0;
}
