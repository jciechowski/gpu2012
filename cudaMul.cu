#include "hip/hip_runtime.h"
/*
  Mnozenie macierzy CUDA,
  Jakub Ciechowski GPU 2012

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "timers.h"

int TILE_WIDTH = 5;

__global__ void tilingMul(int *M, int *N, int *P, int width, int TILE_WIDTH) {
  
  int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
  
  int sum = 0;
  
  for(int k = 0; k < width; k++)
    sum += M[row*width+k] * N[k*width+col];
  
  P[row*width+col] = sum;
  
}

void printMat(int *a, int width) {
  int i,j;
  for(i=0;i<width;i++) 
    for(j=0;j<width;j++)
      printf("%d%c",a[i*width+j],(j == (width-1))?'\n':'\t');

  printf("\n");
}

int* genMatrix(int width) {
  int *a = (int*)calloc(width*width, sizeof(int));
  int i,j;
  for(i=0;i<width;i++)
    for(j=0;j<width;j++)
      a[i*width+j] = rand()%10;
  return a;
}

int *matMul(int *hostA, int *hostB, int width) {
  int *hostC = (int*)malloc(width*width*sizeof(int));

  int *devA, *devB, *devC;
  hipMalloc((void**) &devA, width*width*sizeof(int));
  hipMalloc((void**) &devB, width*width*sizeof(int));
  hipMalloc((void**) &devC, width*width*sizeof(int));
    
  TILE_WIDTH = sqrt(width);
  dim3 blockDim(width/TILE_WIDTH, width/TILE_WIDTH);
  dim3 gridDim(TILE_WIDTH, TILE_WIDTH);
  hipMemcpy(devA, hostA, width*width*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devB, hostB, width*width*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devC, hostC, width*width*sizeof(int), hipMemcpyHostToDevice);
  pTimer zegar = newTimer();
  printf("n = %d\n",width);
  startTimer(zegar);
  tilingMul<<<gridDim, blockDim>>> (devA, devB, devC, width, TILE_WIDTH);
  hipMemcpy(hostC, devC, width*width*sizeof(int), hipMemcpyDeviceToHost);
  stopTimer(zegar);
  printTimer(zegar);
  printf("\n");
    
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  freeTimer(zegar);
  return hostC;
}

int main(int argc, char** argv) {
  int m = 4;
  int *A;
  int *B;
  int *C;
  if(argc > 1) {
    m = atoi(argv[1]);
  }

  A = genMatrix(m);
  /*
    printMat(A,m);
  */
  B = genMatrix(m);
  /*
    printMat(B,m);
  */
  C = matMul(A,B,m*i);

  /*
    printMat(C,m);
  */
    
  return 0;
}
